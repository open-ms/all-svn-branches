#include "hip/hip_runtime.h"
// -*- Mode: C++; tab-width: 2; -*-
// vi: set ts=2:
//
// --------------------------------------------------------------------------
//                   OpenMS Mass Spectrometry Framework
// --------------------------------------------------------------------------
//  Copyright (C) 2003-2010 -- Oliver Kohlbacher, Knut Reinert
//
//  This library is free software; you can redistribute it and/or
//  modify it under the terms of the GNU Lesser General Public
//  License as published by the Free Software Foundation; either
//  version 2.1 of the License, or (at your option) any later version.
//
//  This library is distributed in the hope that it will be useful,
//  but WITHOUT ANY WARRANTY; without even the implied warranty of
//  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
//  Lesser General Public License for more details.
//
//  You should have received a copy of the GNU Lesser General Public
//  License along with this library; if not, write to the Free Software
//  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
//
// --------------------------------------------------------------------------
// $Maintainer: Rene Hussong $
// $Authors: Rene Hussong $
// --------------------------------------------------------------------------

using namespace std;

//**************************
//Uses the sorting code provided by Alan Kaatz
//http://courses.ece.uiuc.edu/ece498/al1/mps/MP5-TopWinners/kaatz/MP5-parallel_sort.zip
//**************************

#include <OpenMS/TRANSFORMATIONS/FEATUREFINDER/IsotopeWaveletCudaKernel.h>
#include <OpenMS/TRANSFORMATIONS/FEATUREFINDER/IsotopeWaveletConstants.h>

#include <iostream>
#include <fstream>
#include <math.h>
#include <vector>
#include <iomanip>

#include <assert.h>
#include <hip/hip_runtime.h>


texture<float,1> trans_intensities_tex, pos_tex, int_tex;
texture<int, 1> sorted_positions_indices_tex;

namespace OpenMS
{
	
	int checkCUDAError(const char *msg)
	{
			hipError_t err = hipGetLastError();
			if( hipSuccess != err) 
			{
					fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
					return (-1);
			};     
			return (0);       
	}

	
	__device__ float isotope_wavelet (float tz1, float mz)
	{
		float fac (-(Constants::LAMBDA_L_0 + Constants::LAMBDA_L_1*mz));
		fac += (tz1-1)*__log2f(-fac)*Constants::ONEOLOG2E - lgammaf(tz1);
			
		return (__sinf((tz1-1)*Constants::WAVELET_PERIODICITY) * __expf(fac));
	}

	__device__ unsigned int getMzPeakCutOffAtMonoPos (float mass, unsigned int  z)
	{
		float mz (mass*z);
		if (mz>Constants::CUT_LAMBDA_BREAK_1_2)
			return(ceil(Constants::CUT_LAMBDA_L_2_A+Constants::CUT_LAMBDA_L_2_B*mz));
		if (mz<Constants::CUT_LAMBDA_BREAK_0_1)
			return(ceil(Constants::CUT_LAMBDA_Q_0_A+Constants::CUT_LAMBDA_Q_0_B*mz+Constants::CUT_LAMBDA_Q_0_C*mz*mz));
		
		return(ceil(Constants::CUT_LAMBDA_Q_1_A+Constants::CUT_LAMBDA_Q_1_B*mz+Constants::CUT_LAMBDA_Q_1_C*mz*mz));
	}
	
	__device__ unsigned int getNumPeakCutOff (float mass, unsigned int  z)
	{
		float mz (mass*z);
		if (mz<Constants::CUT_LAMBDA_BREAK_0_1)
			return(ceil(Constants::CUT_LAMBDA_Q_0_A+Constants::CUT_LAMBDA_Q_0_B*mz+Constants::CUT_LAMBDA_Q_0_C*mz*mz-Constants::IW_QUARTER_NEUTRON_MASS));
		if (mz>Constants::CUT_LAMBDA_BREAK_1_2)
			return(ceil(Constants::CUT_LAMBDA_L_2_A+Constants::CUT_LAMBDA_L_2_B*mz-Constants::IW_QUARTER_NEUTRON_MASS));
		
		return(ceil(Constants::CUT_LAMBDA_Q_1_A+Constants::CUT_LAMBDA_Q_1_B*mz+Constants::CUT_LAMBDA_Q_1_C*mz*mz-Constants::IW_QUARTER_NEUTRON_MASS));
	}

	__device__ unsigned int getNumPeakCutOff (float mz)
	{		
		if (mz<Constants::CUT_LAMBDA_BREAK_0_1)
			return(ceil(Constants::CUT_LAMBDA_Q_0_A+Constants::CUT_LAMBDA_Q_0_B*mz+Constants::CUT_LAMBDA_Q_0_C*mz*mz-Constants::IW_QUARTER_NEUTRON_MASS));
		if (mz>Constants::CUT_LAMBDA_BREAK_1_2)
			return(ceil(Constants::CUT_LAMBDA_L_2_A+Constants::CUT_LAMBDA_L_2_B*mz-Constants::IW_QUARTER_NEUTRON_MASS));
			
		return(ceil(Constants::CUT_LAMBDA_Q_1_A+Constants::CUT_LAMBDA_Q_1_B*mz+Constants::CUT_LAMBDA_Q_1_C*mz*mz-Constants::IW_QUARTER_NEUTRON_MASS));
	}


	__global__ void ConvolutionIsotopeWaveletKernel(float* signal_pos, float* signal_int, const int from_max_to_left, const int from_max_to_right, float* result, 
		const unsigned int charge, const int to_load, const int to_compute, const int size)
	{
		// the device-shared memory storing one data block
		// This is currently hard-coded to 256 points, since we require two 4B floats for each
		// data point, leading to 2kB per block.
		__shared__ float signal_pos_block[Constants::CUDA_EXTENDED_BLOCK_SIZE_MAX];//[BLOCK_SIZE_MAX];
		__shared__ float signal_int_block[Constants::CUDA_EXTENDED_BLOCK_SIZE_MAX];//[BLOCK_SIZE_MAX];

		// load the data from device memory to shared memory. 
		// to distribute the loads as evenly as possible over the threads, each thread loads
		// the data point it will later compute in the output. the first wavelet_length threads
		// will also load the padding to the left of the signal, the last wavelet_length ones will 
		// load the padding to the right
		
		// we will silently ignore the first wavelet_length points in the output; these have to be
		// zero-padded by the calling function. our data organization is as follows: each block computes
		// a part of the output that is block_size-2*wavelet_length points long. For the computation, we
		// require wavelet_length points on the left and on the right so we can put the wavelet on all
		// points even at the boundary.
		//                 left padding,                                                      position of thread
		//                 ignored in output    the points computed by the previous blocks    in block
		int my_data_pos  = from_max_to_left    +  blockIdx.x*to_compute + threadIdx.x;

		int my_local_pos = threadIdx.x + from_max_to_left;

		//every thread with an ID smaller than the number of from_max_to_left loads the additional boundary points
		//at the left end
		if (threadIdx.x < from_max_to_left)
		{
			signal_pos_block[threadIdx.x] = signal_pos[my_data_pos-from_max_to_left];
			signal_int_block[threadIdx.x] = signal_int[my_data_pos-from_max_to_left];
		}
			
		int additional_right_end_loads=0;
		while (my_local_pos + (additional_right_end_loads)*Constants::CUDA_BLOCK_SIZE_MAX < to_load)
		{
			signal_pos_block[my_local_pos+additional_right_end_loads*Constants::CUDA_BLOCK_SIZE_MAX] = signal_pos[my_data_pos+additional_right_end_loads*(Constants::CUDA_BLOCK_SIZE_MAX)];
			signal_int_block[my_local_pos+additional_right_end_loads*Constants::CUDA_BLOCK_SIZE_MAX] = signal_int[my_data_pos+additional_right_end_loads*(Constants::CUDA_BLOCK_SIZE_MAX)];
			++additional_right_end_loads;
		};
		
		//wait until the shared data is loaded completely
		__syncthreads(); 

		if (threadIdx.x >= to_compute || 	my_data_pos - from_max_to_left >= size)
			return;

		float value = 0, boundary = getMzPeakCutOffAtMonoPos(signal_pos_block[my_local_pos], charge)/(float)charge;
		float old=0, c_diff, current, old_pos = (my_local_pos-from_max_to_left-1) > 0 ? signal_pos_block[my_local_pos-from_max_to_left-1] 
			: signal_pos_block[my_local_pos-from_max_to_left]-(signal_pos[size-1]-signal_pos[size-2]); //i.e. min_spacing

		for (int current_conv_pos = my_local_pos-from_max_to_left; 
						current_conv_pos < my_local_pos+from_max_to_right; 
							++current_conv_pos)
		{
			c_diff = signal_pos_block[current_conv_pos]-signal_pos_block[my_local_pos]+Constants::IW_QUARTER_NEUTRON_MASS/(float)charge;

			//Attention! The +1. has nothing to do with the charge, it is caused by the wavelet's formula (tz1).
			current = c_diff > 0 && c_diff <= boundary ? isotope_wavelet(c_diff*charge+1., signal_pos_block[current_conv_pos]*charge)*signal_int_block[current_conv_pos] : 0;
			
			value += 0.5*(current + old)*(signal_pos_block[current_conv_pos]-old_pos);
			
			old = current;
			old_pos = signal_pos_block[current_conv_pos];
		};
		
		result[my_data_pos] = value;
	}

	__global__ void ConvolutionIsotopeWaveletKernelHighRes(float* signal_pos, float* signal_int, const int from_max_to_left, const int from_max_to_right, float* result, 
		const unsigned int charge, const int to_load, const int to_compute, const int size)
	{
		// the device-shared memory storing one data block
		// This is currently hard-coded to 256 points, since we require two 4B floats for each
		// data point, leading to 2kB per block.
		__shared__ float signal_pos_block[Constants::CUDA_EXTENDED_BLOCK_SIZE_MAX];//[BLOCK_SIZE_MAX];
		__shared__ float signal_int_block[Constants::CUDA_EXTENDED_BLOCK_SIZE_MAX];//[BLOCK_SIZE_MAX];

		// load the data from device memory to shared memory. 
		// to distribute the loads as evenly as possible over the threads, each thread loads
		// the data point it will later compute in the output. the first wavelet_length threads
		// will also load the padding to the left of the signal, the last wavelet_length ones will 
		// load the padding to the right
		
		// we will silently ignore the first wavelet_length points in the output; these have to be
		// zero-padded by the calling function. our data organization is as follows: each block computes
		// a part of the output that is block_size-2*wavelet_length points long. For the computation, we
		// require wavelet_length points on the left and on the right so we can put the wavelet on all
		// points even at the boundary.
		//                 left padding,                                                      position of thread
		//                 ignored in output    the points computed by the previous blocks    in block
		int my_data_pos  = from_max_to_left    +  blockIdx.x*to_compute + threadIdx.x;

		int my_local_pos = threadIdx.x + from_max_to_left;

		//every thread with an ID smaller than the number of from_max_to_left loads the additional boundary points
		//at the left end
		if (threadIdx.x < from_max_to_left)
		{
			signal_pos_block[threadIdx.x] = signal_pos[my_data_pos-from_max_to_left];
			signal_int_block[threadIdx.x] = signal_int[my_data_pos-from_max_to_left];
		}
			
		int additional_right_end_loads=0;
		while (my_local_pos + (additional_right_end_loads)*Constants::CUDA_BLOCK_SIZE_MAX < to_load)
		{
			signal_pos_block[my_local_pos+additional_right_end_loads*Constants::CUDA_BLOCK_SIZE_MAX] = signal_pos[my_data_pos+additional_right_end_loads*(Constants::CUDA_BLOCK_SIZE_MAX)];
			signal_int_block[my_local_pos+additional_right_end_loads*Constants::CUDA_BLOCK_SIZE_MAX] = signal_int[my_data_pos+additional_right_end_loads*(Constants::CUDA_BLOCK_SIZE_MAX)];
			++additional_right_end_loads;
		};
		
		//wait until the shared data is loaded completely
		__syncthreads(); 

		if (threadIdx.x >= to_compute || 	my_data_pos - from_max_to_left >= size)
			return;

		float value = 0, boundary = getMzPeakCutOffAtMonoPos(signal_pos_block[my_local_pos], charge)/(float)charge;
		float c_diff, current;

		for (int current_conv_pos = my_local_pos-from_max_to_left; 
						current_conv_pos < my_local_pos+from_max_to_right; 
							++current_conv_pos)
		{
			c_diff = signal_pos_block[current_conv_pos]-signal_pos_block[my_local_pos]+Constants::IW_QUARTER_NEUTRON_MASS/(float)charge;

			//Attention! The +1. has nothing to do with the charge, it is caused by the wavelet's formula (tz1).
			current = c_diff > 0 && c_diff <= boundary ? isotope_wavelet(c_diff*charge+1., signal_pos_block[current_conv_pos]*charge)*signal_int_block[current_conv_pos] : 0;
			
			value += current;
		};
		
		result[my_data_pos] = value;
	}


	__global__ void ConvolutionIsotopeWaveletKernelTexture(const int from_max_to_left, const int from_max_to_right, float* result, 
		const unsigned int charge, const int size)
	{
		int my_data_pos  = from_max_to_left    +  blockIdx.x*blockDim.x + threadIdx.x;

		if (my_data_pos - from_max_to_left >= size)
			return;

		float value = 0, boundary = getMzPeakCutOffAtMonoPos(tex1Dfetch(pos_tex, my_data_pos), charge)/(float)charge;
		float old=0, c_diff, current, old_pos = (my_data_pos-from_max_to_left-1) > 0 ? tex1Dfetch(pos_tex, my_data_pos-from_max_to_left-1) 
				: tex1Dfetch(pos_tex,my_data_pos-from_max_to_left)-(tex1Dfetch(pos_tex, size-1)-tex1Dfetch(pos_tex, size-2)); //i.e. min_spacing

		for (int current_conv_pos = my_data_pos-from_max_to_left; 
						current_conv_pos < my_data_pos+from_max_to_right; 
							++current_conv_pos)
		{
			c_diff =  tex1Dfetch(pos_tex, current_conv_pos)- tex1Dfetch(pos_tex, my_data_pos)+Constants::IW_QUARTER_NEUTRON_MASS/(float)charge;

			//Attention! The +1. has nothing to do with the charge, it is caused by the wavelet's formula (tz1).
			current = c_diff > 0 && c_diff <= boundary ? isotope_wavelet(c_diff*charge+1., tex1Dfetch(pos_tex, current_conv_pos)*charge)*tex1Dfetch(int_tex, current_conv_pos) : 0;
			value += 0.5*(current + old)*(tex1Dfetch(pos_tex, current_conv_pos)-old_pos);//current;
			
			old = current;
			old_pos = tex1Dfetch(pos_tex, current_conv_pos);
		};

		result[my_data_pos] = value;
	}


	__global__ void ConvolutionIsotopeWaveletKernelTextureHighRes(const int from_max_to_left, const int from_max_to_right, float* result, 
		const unsigned int charge, const int size)
	{
		int my_data_pos  = from_max_to_left    +  blockIdx.x*blockDim.x + threadIdx.x;

		if (my_data_pos - from_max_to_left >= size)
			return;

		float value = 0, boundary = getMzPeakCutOffAtMonoPos(tex1Dfetch(pos_tex, my_data_pos), charge)/(float)charge;
		float c_diff, current;
 
		for (int current_conv_pos = my_data_pos-from_max_to_left; 
						current_conv_pos < my_data_pos+from_max_to_right; 
							++current_conv_pos)
		{
			c_diff =  tex1Dfetch(pos_tex, current_conv_pos)- tex1Dfetch(pos_tex, my_data_pos)+Constants::IW_QUARTER_NEUTRON_MASS/(float)charge;

			//Attention! The +1. has nothing to do with the charge, it is caused by the wavelet's formula (tz1).
			current = c_diff > 0 && c_diff <= boundary ? isotope_wavelet(c_diff*charge+1., tex1Dfetch(pos_tex, current_conv_pos)*charge)*tex1Dfetch(int_tex, current_conv_pos) : 0;
			value += current;
			
		};

		result[my_data_pos] = value;
	}

	__global__ void getDerivatives (float* spec, float* spec_pos, float* fwd2, const int size, float* intensities_dev)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
	
		if ((i+2>=size && i<size) || i==0)
		{
			fwd2[i] = 0;
			return;
		};
	
		float share = spec[i+1], share_pos = spec_pos[i+1];
		float bwd = (share-spec[i])/(share_pos-spec_pos[i]);
		float fwd = (spec[i+2]-share)/(spec_pos[i+2]-share_pos);

		if (bwd>=0 && fwd<=0 && share <= intensities_dev[i+1])
		{
			fwd2[i+1] = spec[i+1];
		};
	}
	
	__global__ void getDerivativesHighRes (float* spec, float* spec_pos, float* fwd2, const int size, float* intensities_dev)
	{
		int i = threadIdx.x + blockIdx.x * blockDim.x;
	
		if ((i+2>=size && i<size) || i==0)
		{
			fwd2[i] = 0;
			return;
		};
	
		float share = spec[i+1], share_pos = spec_pos[i+1];
		float bwd = (share-spec[i])/(share_pos-spec_pos[i]);
		float fwd = (spec[i+2]-share)/(spec_pos[i+2]-share_pos);

		if (bwd>=0 && fwd<=0)
		{
			fwd2[i+1] = spec[i+1];
		};
	}


	void deriveOnDevice (float* spec, float* spec_pos, float* fwd, const int size, float* intensities_dev, bool hr_data)
	{
		dim3 blockDim (Constants::CUDA_BLOCK_SIZE_MAX);
		dim3 gridDim ((int)(ceil)(size/(float)Constants::CUDA_BLOCK_SIZE_MAX));
		if (hr_data)
		{
			getDerivativesHighRes<<<gridDim, blockDim>>> (spec, spec_pos, fwd, size, intensities_dev);
		}
		else
		{
			getDerivatives<<<gridDim, blockDim>>> (spec, spec_pos, fwd, size, intensities_dev);
		}
		hipDeviceSynchronize();
		checkCUDAError("deriveOnDevice");
	}


	void getExternalCudaTransforms (dim3 dimGrid, dim3 dimBlock, float* positions_dev, float* intensities_dev, int from_max_to_left, int from_max_to_right, float* result_dev, 
		const int charge, const int to_load, const int to_compute, const int size, float* fwd2, bool hr_data) 
	{
		if (to_load < Constants::CUDA_EXTENDED_BLOCK_SIZE_MAX)
		{	
			if (hr_data)
			{
				ConvolutionIsotopeWaveletKernelHighRes<<<dimGrid,dimBlock>>> (positions_dev, intensities_dev, from_max_to_left, from_max_to_right, result_dev, charge, to_load, to_compute, size);
			}
			else
			{
				ConvolutionIsotopeWaveletKernel<<<dimGrid,dimBlock>>> (positions_dev, intensities_dev, from_max_to_left, from_max_to_right, result_dev, charge, to_load, to_compute, size);
			}
			hipDeviceSynchronize();
			checkCUDAError("ConvolutionIsotopeWaveletKernel");
			deriveOnDevice (result_dev, positions_dev, fwd2, size, intensities_dev, hr_data);
		}
		else
		{
                	std::cout << "Must use texture instead of shared memory. To load: " << to_load << "\t" << Constants::CUDA_EXTENDED_BLOCK_SIZE_MAX << std::endl;
			dimBlock = dim3(Constants::CUDA_TEXTURE_THREAD_LIMIT);
			dimGrid = dim3((int)ceil(size/(float)dimBlock.x));
			hipBindTexture(0, int_tex, intensities_dev, (size+from_max_to_left+from_max_to_right)*sizeof(float));
			hipBindTexture(0, pos_tex, positions_dev, (size+from_max_to_left+from_max_to_right)*sizeof(float));

			if (hr_data)
			{
				ConvolutionIsotopeWaveletKernelTextureHighRes<<<dimGrid,dimBlock>>> (from_max_to_left, from_max_to_right, result_dev, charge, size);
			}
			else
			{
				ConvolutionIsotopeWaveletKernelTexture<<<dimGrid,dimBlock>>> (from_max_to_left, from_max_to_right, result_dev, charge, size);
			}
			hipDeviceSynchronize();
			checkCUDAError("ConvolutionIsotopeWaveletKernelTexture");
			deriveOnDevice (result_dev, positions_dev, fwd2, size, intensities_dev, hr_data);
			
			hipUnbindTexture(int_tex);
			hipUnbindTexture(pos_tex);
		};
	}


	__device__ inline void swap(float &a, float &b, int &c, int &d) 
	{
			float tmp (a);
			a = b;
			b = tmp;
				
			int tmp2 (c);
			c = d;
			d = tmp2;
	}

	__global__ void sharedMemMerge(float *array, int *pos, int k) {

			__shared__ float shmem[Constants::CUDA_ELEMENTS_MERGE];
			__shared__ int posshmem[Constants::CUDA_ELEMENTS_MERGE];

			int tmp = blockIdx.x * Constants::CUDA_ELEMENTS_MERGE + threadIdx.x;

			float data = array[tmp];
			float data2 = array[tmp + (Constants::CUDA_ELEMENTS_MERGE / 2)];

			float data3 = array[tmp + Constants::CUDA_THREADS_MERGE];
			float data4 = array[tmp + Constants::CUDA_THREADS_MERGE + (Constants::CUDA_ELEMENTS_MERGE / 2)];
			
			int posdata = pos[tmp];
			int posdata2 = pos[tmp + (Constants::CUDA_ELEMENTS_MERGE / 2)];

			int posdata3 = pos[tmp + Constants::CUDA_THREADS_MERGE];
			int posdata4 = pos[tmp + Constants::CUDA_THREADS_MERGE + (Constants::CUDA_ELEMENTS_MERGE / 2)];

			int dir = k & (blockIdx.x * (Constants::CUDA_ELEMENTS_MERGE));


			if (dir == 0) {
					if (data > data2) {  // ascending
							shmem[threadIdx.x] = data2;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = data;
							posshmem[threadIdx.x] = posdata2;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = posdata;
					} else {
							shmem[threadIdx.x] = data;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = data2;
							posshmem[threadIdx.x] = posdata;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = posdata2;
					}

					if (data3 > data4) {  // ascending
							shmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = data4;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = data3;
							posshmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = posdata4;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = posdata3;
					} else {
							shmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = data3;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = data4;
							posshmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = posdata3;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = posdata4;
					}
			} else {
					if (data < data2) {  // descending
							shmem[threadIdx.x] = data2;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = data;
							posshmem[threadIdx.x] = posdata2;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = posdata;
					} else {
							shmem[threadIdx.x] = data;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = data2;							
							posshmem[threadIdx.x] = posdata;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2)] = posdata2;
					}

					if (data3 < data4) {  // descending
							shmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = data4;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = data3;
							posshmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = posdata4;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = posdata3;
					} else {
							shmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = data3;
							shmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = data4;							
							posshmem[threadIdx.x + Constants::CUDA_THREADS_MERGE] = posdata3;
							posshmem[threadIdx.x + (Constants::CUDA_ELEMENTS_MERGE / 2) + Constants::CUDA_THREADS_MERGE] = posdata4;
					}
			}



			int j = 256, s = Constants::CUDA_MERGE_NUM >> 2; 


			int x = threadIdx.x + (s & threadIdx.x);
			int y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);							
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}
			
			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);	
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			j >>= 1;
			s >>= 1;


			x = threadIdx.x + (s & threadIdx.x);
			y = x + j;
			__syncthreads();

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}

			x = (threadIdx.x + Constants::CUDA_THREADS_MERGE) + ((threadIdx.x + Constants::CUDA_THREADS_MERGE) & s);
			y = x + j;

			if (dir == 0) {
					if (shmem[x] > shmem[y]) {  // ascending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			} else {
					if (shmem[x] < shmem[y]) {  // descending
							swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
					}
			}


			__syncthreads();

			int i = blockIdx.x * Constants::CUDA_ELEMENTS_MERGE + threadIdx.x;
			array[i] = shmem[threadIdx.x];
			array[i + (Constants::CUDA_ELEMENTS_MERGE / 4)] = shmem[(Constants::CUDA_ELEMENTS_MERGE / 4) + threadIdx.x];
			array[i + (Constants::CUDA_ELEMENTS_MERGE / 2)] = shmem[(Constants::CUDA_ELEMENTS_MERGE / 2) + threadIdx.x];
			array[i + (3 * Constants::CUDA_ELEMENTS_MERGE / 4)] = shmem[(3 * Constants::CUDA_ELEMENTS_MERGE / 4) + threadIdx.x];

			pos[i] = posshmem[threadIdx.x];
			pos[i + (Constants::CUDA_ELEMENTS_MERGE / 4)] = posshmem[(Constants::CUDA_ELEMENTS_MERGE / 4) + threadIdx.x];
			pos[i + (Constants::CUDA_ELEMENTS_MERGE / 2)] = posshmem[(Constants::CUDA_ELEMENTS_MERGE / 2) + threadIdx.x];
			pos[i + (3 * Constants::CUDA_ELEMENTS_MERGE / 4)] = posshmem[(3 * Constants::CUDA_ELEMENTS_MERGE / 4) + threadIdx.x];
	}




	__global__ void mergeArray(float *array, int* pos, int j, int k, int s) {
			int tmp = (blockIdx.x * Constants::CUDA_THREADS_GL);
			int x = tmp +  threadIdx.x + (tmp & s);
			j += x;

			float data1 = array[x];
			float data2 = array[j];

			if ((x & k) == 0) {    // ascending
					if (data1 > data2) {
							swap(array[x], array[j], pos[x], pos[j]);
					}
			} else {                // descending
					if (data1 < data2) {
							swap(array[x], array[j], pos[x], pos[j]);
					}
			}
	}




	__global__ void sharedMemSort(float2 *array, int2 *pos) 
	{
			__shared__ float shmem[Constants::CUDA_ELEMENTS_SORT];
			__shared__ int posshmem[Constants::CUDA_ELEMENTS_SORT];

			float2 data = array[blockIdx.x * (Constants::CUDA_ELEMENTS_SORT / 2) + threadIdx.x];
			int2 posdata = pos[blockIdx.x * (Constants::CUDA_ELEMENTS_SORT / 2) + threadIdx.x];


			if ( (threadIdx.x & 1) == 0) {
					if (data.x > data.y) {  // ascending
							shmem[2 * threadIdx.x] = data.y;
							shmem[2 * threadIdx.x + 1] = data.x;
							posshmem[2 * threadIdx.x] = posdata.y;
							posshmem[2 * threadIdx.x + 1] = posdata.x;
					} else {
							shmem[2 * threadIdx.x] = data.x;
							shmem[2 * threadIdx.x + 1] = data.y;									
							posshmem[2 * threadIdx.x] = posdata.x;
							posshmem[2 * threadIdx.x + 1] = posdata.y;							
					}
			} else {
					if (data.x < data.y) {  // descending
							shmem[2 * threadIdx.x] = data.y;
							shmem[2 * threadIdx.x + 1] = data.x;							
							posshmem[2 * threadIdx.x] = posdata.y;
							posshmem[2 * threadIdx.x + 1] = posdata.x;

					} else {
							shmem[2 * threadIdx.x] = data.x;
							shmem[2 * threadIdx.x + 1] = data.y;							
							posshmem[2 * threadIdx.x] = posdata.x;
							posshmem[2 * threadIdx.x + 1] = posdata.y;
					}
			}


			for (int k = 4, r = 0xFFFFFFFC; k <= (Constants::CUDA_ELEMENTS_SORT / 2); k *= 2, r <<= 1) {

					for (int j = k >> 1, s = r >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							
							__syncthreads();
							
							if ((x & k) == 0) {

									if (shmem[x] > shmem[y]) {  // ascending
											swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
									}
							} else {
									if (shmem[x] < shmem[y]) {  // descending
											swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
									}
							}
					}
			}


			data = array[blockIdx.x * (Constants::CUDA_ELEMENTS_SORT / 2) + Constants::CUDA_THREADS_SORT + threadIdx.x];
			posdata = pos[blockIdx.x * (Constants::CUDA_ELEMENTS_SORT / 2) + Constants::CUDA_THREADS_SORT + threadIdx.x];
			float* shmem2 = &shmem[Constants::CUDA_ELEMENTS_SORT / 2];
			int* posshmem2 = &posshmem[Constants::CUDA_ELEMENTS_SORT / 2];
			
			__syncthreads();

			if ( (threadIdx.x & 1) == 0) {
					if (data.x > data.y) {  // ascending
							shmem2[2 * threadIdx.x] = data.y;
							shmem2[2 * threadIdx.x + 1] = data.x;
							posshmem2[2 * threadIdx.x] = posdata.y;
							posshmem2[2 * threadIdx.x + 1] = posdata.x;
					} else {
							shmem2[2 * threadIdx.x] = data.x;
							shmem2[2 * threadIdx.x + 1] = data.y;							
							posshmem2[2 * threadIdx.x] = posdata.x;
							posshmem2[2 * threadIdx.x + 1] = posdata.y;
					}
			} else {
					if (data.x < data.y) {  // descending
							shmem2[2 * threadIdx.x] = data.y;
							shmem2[2 * threadIdx.x + 1] = data.x;
							posshmem2[2 * threadIdx.x] = posdata.y;
							posshmem2[2 * threadIdx.x + 1] = posdata.x;
					} else {
							shmem2[2 * threadIdx.x] = data.x;
							shmem2[2 * threadIdx.x + 1] = data.y;							
							posshmem2[2 * threadIdx.x] = posdata.x;
							posshmem2[2 * threadIdx.x + 1] = posdata.y;
					}
			}


			for (int k = 4, r = 0xFFFFFFFC; k <= (Constants::CUDA_ELEMENTS_SORT / 2); k *= 2, r <<= 1) {

					for (int j = k >> 1, s = r >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							__syncthreads();

							if ((x & k) == 0) {
									if (shmem2[x] < shmem2[y]) {  // descending
											swap(shmem2[x], shmem2[y], posshmem2[x], posshmem2[y]);	
									}
							} else {
									if (shmem2[x] > shmem2[y]) {  // ascending
											swap(shmem2[x], shmem2[y], posshmem2[x], posshmem2[y]);
									}
							}
					}
			}


			if ((blockIdx.x & 1) == 0) {

					for (int j = Constants::CUDA_ELEMENTS_SORT / 2, s = Constants::CUDA_SORT_NUM >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							__syncthreads();

							if (shmem[x] > shmem[y]) {  // ascending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}

							x = (threadIdx.x + Constants::CUDA_THREADS_SORT) + ((threadIdx.x + Constants::CUDA_THREADS_SORT) & s);
							y = x + j;
			
							if (shmem[x] > shmem[y]) {  // ascending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}
					}
			
			} else {
			
					for (int j = Constants::CUDA_ELEMENTS_SORT / 2, s = Constants::CUDA_SORT_NUM >> 1; j > 0; j >>= 1, s >>= 1) {

							int x = threadIdx.x + (threadIdx.x & s);
							int y = x + j;
							__syncthreads();

							if (shmem[x] < shmem[y]) {  // descending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}

							x = (threadIdx.x + Constants::CUDA_THREADS_SORT) + ((threadIdx.x + Constants::CUDA_THREADS_SORT) & s);
							y = x + j;
			
							if (shmem[x] < shmem[y]) {  // descending
									swap(shmem[x], shmem[y], posshmem[x], posshmem[y]);
							}
					}
			
			}


			__syncthreads();

			int i = blockIdx.x * Constants::CUDA_ELEMENTS_SORT + threadIdx.x;
			((float*)array)[i] = shmem[threadIdx.x];
			((float*)array)[i + (Constants::CUDA_ELEMENTS_SORT / 4)] = shmem[(Constants::CUDA_ELEMENTS_SORT / 4) + threadIdx.x];
			((float*)array)[i + (Constants::CUDA_ELEMENTS_SORT / 2)] = shmem[(Constants::CUDA_ELEMENTS_SORT / 2) + threadIdx.x];
			((float*)array)[i + (3 * Constants::CUDA_ELEMENTS_SORT / 4)] = shmem[(3 * Constants::CUDA_ELEMENTS_SORT / 4) + threadIdx.x];

			((int*)pos)[i] = posshmem[threadIdx.x];
			((int*)pos)[i + (Constants::CUDA_ELEMENTS_SORT / 4)] = posshmem[(Constants::CUDA_ELEMENTS_SORT / 4) + threadIdx.x];
			((int*)pos)[i + (Constants::CUDA_ELEMENTS_SORT / 2)] = posshmem[(Constants::CUDA_ELEMENTS_SORT / 2) + threadIdx.x];
			((int*)pos)[i + (3 * Constants::CUDA_ELEMENTS_SORT / 4)] = posshmem[(3 * Constants::CUDA_ELEMENTS_SORT / 4) + threadIdx.x];
	}


	__global__ void findCutOffIndex (float* array, int* cut_off_index)
	{
		int my_index = blockIdx.x*blockDim.x + threadIdx.x;

		if (my_index+1 >= gridDim.x*blockDim.x)
		{
			//printf ("returning\n");
			return;
		};
		float first = array[my_index], second = array[my_index+1];
		//printf ("my_index: %i\t%f\t%f\n", my_index, first, second);

		if (first <=0 && second > 0)
		{
			//printf ("writing: %i",  my_index+1);
			*cut_off_index = my_index+1;
		};
	};


	int sortOnDevice(float *array, int* pos_indices, int numElements, int padding)
	{
		dim3 dimGridSharedMemSort((numElements / Constants::CUDA_ELEMENTS_SORT) - (padding / Constants::CUDA_ELEMENTS_SORT), 1, 1);
    dim3 dimBlockSharedMemSort(Constants::CUDA_THREADS_SORT, 1, 1);

    dim3 dimGridMergeArray(numElements / (Constants::CUDA_THREADS_GL * Constants::CUDA_ELEMENTS_GL), 1, 1);
    dim3 dimBlockMergeArray(Constants::CUDA_THREADS_GL, 1, 1);

    dim3 dimGridSharedMemMerge(numElements / Constants::CUDA_ELEMENTS_MERGE, 1, 1);
    dim3 dimBlockSharedMemMerge(Constants::CUDA_THREADS_MERGE, 1, 1);

		sharedMemSort<<<dimGridSharedMemSort, dimBlockSharedMemSort>>>(((float2*)array) + ((padding / Constants::CUDA_ELEMENTS_SORT) * (Constants::CUDA_ELEMENTS_SORT / 2)), 
			((int2*)pos_indices) + ((padding / Constants::CUDA_ELEMENTS_SORT) * (Constants::CUDA_ELEMENTS_SORT / 2)));

    for (int k = (Constants::CUDA_ELEMENTS_SORT << 1), r = (int)(Constants::CUDA_SORT_NUM << 1); k <= numElements; k *= 2, r <<= 1) 
		{

        for (int j = k / 2, s = r >> 1; j > (Constants::CUDA_ELEMENTS_MERGE / 2); j >>= 1, s >>= 1) 
				{
            mergeArray<<<dimGridMergeArray, dimBlockMergeArray>>>(array, pos_indices, j, k, s);
        }

        sharedMemMerge<<<dimGridSharedMemMerge, dimBlockSharedMemMerge>>>(array, pos_indices, k);
    }		
		hipDeviceSynchronize();
		checkCUDAError("sortOnDevice");

		int num_threads = Constants::CUDA_BLOCK_SIZE_MAX;
		while (numElements < num_threads && num_threads > 1)
		{
			num_threads /= 2;
		};

		if (num_threads == 1) //this case should never happen
		{
			return (0);
		}; 

		dim3 dimGrid (numElements/num_threads);
		dim3 dimBlock (num_threads);

		void* dev_cut_off_index;
		hipMalloc (&dev_cut_off_index, sizeof(int));
		hipMemset (dev_cut_off_index, -1, sizeof(int));

		findCutOffIndex<<<dimGrid, dimBlock>>> (array, (int*)dev_cut_off_index);
		hipDeviceSynchronize();
		checkCUDAError("findCutoffIndex");
		int cut_off_index=-1;
		hipMemcpy (&cut_off_index, dev_cut_off_index, sizeof(int), hipMemcpyDeviceToHost);

		return (cut_off_index);
	}


	extern __shared__ float external_shared [];
	__global__ void scoreIndividuals (float* scores, const int overall_size, const int c, const int offset,  const int write_offset, const float ampl_cutoff)
	{		
		int v = threadIdx.x;
		int ref_index = tex1Dfetch (sorted_positions_indices_tex, blockIdx.x+offset);
		
		//printf ("my_index: %i\n", v);
		//printf ("ref_index: %i\n", ref_index); 	
	
		__shared__ int peak_cutoff, optimal_block_dim;
		__shared__ float seed_mz;
		float* c_scores = (float*) &external_shared[0]; 
		
		if (v==0)
		{	
			seed_mz = tex1Dfetch(pos_tex, ref_index);
			//printf ("Scoring: %f\n", seed_mz);

			peak_cutoff = getNumPeakCutOff(seed_mz, c+1);	
			optimal_block_dim = 4*(peak_cutoff-1) -1;
			//optimal_block_dim = 2*(peak_cutoff-1);
		};

		__syncthreads();
		if (v < optimal_block_dim)
		{
			float my_mz, l_pos, l_intens; int l_index;
			my_mz = seed_mz-((peak_cutoff-1)*Constants::IW_NEUTRON_MASS-(v+1)*Constants::IW_HALF_NEUTRON_MASS)/((float)(c+1));
			//my_mz = seed_mz+v*Constants::IW_HALF_NEUTRON_MASS/((float)(c+1));

			l_index = ref_index;
			if (my_mz > seed_mz)
			{ 
				while (l_index < overall_size && tex1Dfetch(pos_tex, l_index++) < my_mz) 
				{ 
				};
				if (l_index<overall_size)
				{
					l_index -= 2;
				};
			}
			else
			{
				while (l_index >= 0 && tex1Dfetch(pos_tex,l_index--) > my_mz) 
				{							
				};
				if (l_index >=0)
				{
					++l_index;
				};					
			};

			if (l_index >=0  && l_index+1 < overall_size)
			{	
				l_pos = tex1Dfetch(pos_tex, l_index);
				l_intens = tex1Dfetch(trans_intensities_tex, l_index);
				c_scores[v] = l_intens + ( tex1Dfetch(trans_intensities_tex, l_index+1)-l_intens ) / (tex1Dfetch(pos_tex, l_index+1) - l_pos) * (my_mz - l_pos); 				
				//printf ("Scoring: %f\t\t%f\t%f\t%f\t%f\t%f\n",  seed_mz, my_mz, tex1Dfetch(pos_tex, l_index+1), l_pos, tex1Dfetch(trans_intensities_tex, l_index+1), l_intens);
			}
			else
			{
				c_scores[v]=INT_MIN;
			};
		};

		__syncthreads();

		//It has been test that an advanced reduction scheme does not offer
		//any performance advantages in our case; so we use the greedy way here ...	
		
		
		 //It has been test that an advanced reduction scheme does not offer
                //any performance advantages in our case; so we use the greedy way here ...
             /*   if (v==0)
                {
                        float final_score = 0, mid_val=0, l_score=0;
                        int minus = -1, i;
                        for (i=0; i<(int)ceil(optimal_block_dim/2.); ++i)
                        {
                                if (c_scores[i] != INT_MIN)
                                {
                                        final_score += minus*c_scores[i];
                                };
                                minus *=-1;
                        };

                        l_score = final_score;
                        mid_val = c_scores[i];

                        for (; i<optimal_block_dim && c_scores[i] != INT_MIN; ++i)
                        {
                                final_score += minus*c_scores[i];
                                minus *=-1;
                        };

                        if (!(l_score <=0 || final_score-l_score-mid_val <= 0 || final_score-mid_val <= ampl_cutoff))
                        {
                                scores[blockIdx.x+write_offset] = final_score;
                        };
                        //printf ("blockid: %i\t%i\n", blockIdx.x, write_offset);
                        //printf("final_score: %f\t\t%f\n", seed_mz, final_score);
                };*/

		
		
		__shared__ float mid_val, l_score, r_score;
		if (v==0)
		{
			l_score=0; mid_val=0;
			int minus = -1; int i;
			for (i=0; i<(int)floor(optimal_block_dim/2.); ++i)
			{
				if (c_scores[i] != INT_MIN)
				{
					//if (trunc(seed_mz*100) == 80442) printf("l: %f\n", c_scores[i]); 
					
					l_score += minus*c_scores[i];
				};
				minus *=-1;
			};
			mid_val = c_scores[i];
			//if (trunc(seed_mz*100) == 80442) printf("m: %f\n", c_scores[i]);
		};
		if(v==1)
		{
			r_score=0;
			int minus = -1;
			for (int i=(int)floor(optimal_block_dim/2.)+1; i<optimal_block_dim && c_scores[i] != INT_MIN; ++i)
			{
				//if (trunc(seed_mz) == 491) printf("r: %f\n", minus*c_scores[i]);

				r_score += minus*c_scores[i];
				minus *=-1;
			};
		};

		__syncthreads();

		if(v==0)
		{
			/*if (trunc(seed_mz) == 491) 
			{
				printf("final_score: %f\t\t%f\t%f\t%f\n", seed_mz,  l_score, mid_val, r_score);
			};*/
	
			if (l_score + r_score <= 0)
			{
			}

			if (!(l_score <=0 || r_score <= 0 || l_score + r_score <= ampl_cutoff))
			{
				scores[blockIdx.x+write_offset] = l_score + r_score + mid_val;	
			}
			else
			{
				if (l_score + r_score <= ampl_cutoff && l_score + r_score > 0)
				{
					scores[blockIdx.x+write_offset]=-1000;
				}
			};
		};
	};

	
	void scoreOnDevice (int* sorted_positions_indices, float* trans_intensities, float* pos, float* scores, 
		const int c, const int num_of_scores, const int overall_size, const unsigned int max_peak_cutoff, const float ampl_cutoff)
	{
		int theo_block_dim = 4*(max_peak_cutoff-1)-1; //the number of scoring points per candidates, due to numerical reasons we increse max_peak_cutoff by one
		dim3 blockDim (theo_block_dim); //the number of scoring points per candidates

		hipBindTexture(0, trans_intensities_tex, trans_intensities, overall_size*sizeof(float));
		hipBindTexture(0, pos_tex, pos, overall_size*sizeof(float));
		hipBindTexture(0, sorted_positions_indices_tex, sorted_positions_indices, overall_size*sizeof(int));	
		size_t offset = overall_size - num_of_scores;

		//printf ("num_of_scores: %i\n", num_of_scores);
		//printf ("overall_size: %i\n", overall_size);
		dim3 gridDim (Constants::CUDA_BLOCKS_PER_GRID_MAX);
		int counts=0, c_size = num_of_scores;

		while ((c_size -= Constants::CUDA_BLOCKS_PER_GRID_MAX) > 0)
		{		
			scoreIndividuals<<<gridDim, blockDim, blockDim.x*sizeof(float)>>> (scores, overall_size, c, 
				counts*Constants::CUDA_BLOCKS_PER_GRID_MAX+offset, counts*Constants::CUDA_BLOCKS_PER_GRID_MAX, ampl_cutoff);	
			++counts;
		};

		if ((c_size += Constants::CUDA_BLOCKS_PER_GRID_MAX) > 0)
		{
			gridDim = dim3 (c_size);
			scoreIndividuals<<<gridDim, blockDim, blockDim.x*sizeof(float)+2*sizeof(int)+sizeof(float)>>> (scores, overall_size, c, 
				counts*Constants::CUDA_BLOCKS_PER_GRID_MAX+offset, counts*Constants::CUDA_BLOCKS_PER_GRID_MAX, ampl_cutoff);		
		};
		
		hipDeviceSynchronize();
		checkCUDAError("scoreOnDevice");
		
		hipUnbindTexture (trans_intensities_tex);
		hipUnbindTexture (pos_tex);
		hipUnbindTexture (sorted_positions_indices_tex);
	}

}	
